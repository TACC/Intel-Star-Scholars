#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <sys/sysinfo.h>
#include <sys/time.h>

#define INDEX(N,i,j) (i*N + j)

unsigned long long rdtsc(void)
{
    unsigned long hi, lo;
    __asm__ __volatile__ ("xorl %%eax, %%eax \n  cpuid" ::: "%eax", "%ebx", "%ecx", "%edx");
    __asm__ __volatile__ ("rdtsc" : "=a"(lo), "=d"(hi));
    return ( (unsigned long long)lo)|( ((unsigned long long)hi)<<32 );
}

static inline unsigned long long int GetTickCount()
{
#ifdef WIN32
    /* TODO find similar function on Windows */
#else
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return tp.tv_sec*1000+tp.tv_usec/1000;
}
#endif

void Calibrate(unsigned long long int *ClkPerSec,double NSecClk)
{
    unsigned long long int start,stop,diff;
    unsigned long long int starttick,stoptick,difftick;

    stoptick = GetTickCount();
    while(stoptick == (starttick=GetTickCount()));

    start = rdtsc();
    while((stoptick=GetTickCount())<(starttick+500));
    stop  = rdtsc();

    diff = stop-start;
    difftick = stoptick-starttick;

    *ClkPerSec = ( diff * (unsigned long long int)1000 )/ (unsigned long long int)(difftick);
    NSecClk = (double)1000000000 / (double)(__int64_t)*ClkPerSec;
}

// Kernel to compute the 5-point stencil and accumulate the norm
__global__ void stencil_kernel(const float *Mat_A, float *Mat_Stencil, int N, int M, float *FNorm) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (row < N-1 && col < M-1) {
        float stencil_value = 4.0f * Mat_A[row * M + col]
                             - Mat_A[(row - 1) * M + col]
                             - Mat_A[(row + 1) * M + col]
                             - Mat_A[row * M + (col - 1)]
                             - Mat_A[row * M + (col + 1)];
        Mat_Stencil[(row - 1) * (M - 2) + (col - 1)] = stencil_value;
        atomicAdd(FNorm, stencil_value * stencil_value);
    }
}



