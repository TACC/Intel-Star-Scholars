#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <sys/sysinfo.h>
#include <sys/time.h>

#define INDEX(N,i,j) (i*N + j)

unsigned long long rdtsc(void)
{
    unsigned long hi, lo;
    __asm__ __volatile__ ("xorl %%eax, %%eax \n  cpuid" ::: "%eax", "%ebx", "%ecx", "%edx");
    __asm__ __volatile__ ("rdtsc" : "=a"(lo), "=d"(hi));
    return ( (unsigned long long)lo)|( ((unsigned long long)hi)<<32 );
}

static inline unsigned long long int GetTickCount()
{
#ifdef WIN32
    /* TODO find similar function on Windows */
#else
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return tp.tv_sec*1000+tp.tv_usec/1000;
}
#endif

void Calibrate(unsigned long long int *ClkPerSec,double NSecClk)
{
    unsigned long long int start,stop,diff;
    unsigned long long int starttick,stoptick,difftick;

    stoptick = GetTickCount();
    while(stoptick == (starttick=GetTickCount()));

    start = rdtsc();
    while((stoptick=GetTickCount())<(starttick+500));
    stop  = rdtsc();

    diff = stop-start;
    difftick = stoptick-starttick;

    *ClkPerSec = ( diff * (unsigned long long int)1000 )/ (unsigned long long int)(difftick);
    NSecClk = (double)1000000000 / (double)(__int64_t)*ClkPerSec;
}

// Kernel to compute the 5-point stencil and accumulate the norm
__global__ void stencil_kernel(const float *Mat_A, float *Mat_Stencil, int N, int M, float *FNorm) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (row < N-1 && col < M-1) {
        float stencil_value = 4.0f * Mat_A[row * M + col]
                             - Mat_A[(row - 1) * M + col]
                             - Mat_A[(row + 1) * M + col]
                             - Mat_A[row * M + (col - 1)]
                             - Mat_A[row * M + (col + 1)];
        Mat_Stencil[(row - 1) * (M - 2) + (col - 1)] = stencil_value;
        atomicAdd(FNorm, stencil_value * stencil_value);
    }
}

// Kernel to compute sqrt of FNorm **once**
__global__ void compute_sqrt(float *FNorm) {
    *FNorm = sqrtf(*FNorm);  // Perform single square root
}

// Kernel to normalize the matrix
__global__ void normalize_kernel(float *Mat_A, const float *Mat_Stencil, float *FNorm, int N, int M) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (row < N - 1 && col < M - 1) {
        Mat_A[row * M + col] = Mat_Stencil[(row - 1) * (M - 2) + (col - 1)] / *FNorm;
    }
}

int main(int argc, char *argv[]) {

    int loops = atoi(argv[1]);
    int N = atoi(argv[2]);
    int M = atoi(argv[3]);

    unsigned long long int ClkPerSec;
    double NSecClk;

    std::vector<float> Mat_A(N * M, 1.0f);
    std::vector<float> Mat_Stencil((N - 2) * (M - 2), 0.0f);
    float FNorm = 0.0f;
    double elapsed_time[10], Average = 0.0;

    Calibrate(&ClkPerSec, NSecClk);

    // Allocate device memory
    float *d_Mat_A, *d_Mat_Stencil, *d_FNorm;
    hipMalloc(&d_Mat_A, N * M * sizeof(float));
    hipMalloc(&d_Mat_Stencil, (N - 2) * (M - 2) * sizeof(float));
    hipMalloc(&d_FNorm, sizeof(float));


    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    for (int count = 0; count < loops; count++) {

        if(FNorm == 0.0f)
          FNorm = 1.0f;
        else
          FNorm = 1.0f;
        unsigned long long start = rdtsc();
        hipMemcpy(d_FNorm, &FNorm, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Mat_A, Mat_A.data(), N * M * sizeof(float), hipMemcpyHostToDevice);


        // Launch the stencil kernel
        stencil_kernel<<<numBlocks, threadsPerBlock>>>(d_Mat_A, d_Mat_Stencil, N, M, d_FNorm);
        hipDeviceSynchronize();

        // Perform single sqrt calculation
        compute_sqrt<<<1, 1>>>(d_FNorm);  // Single thread kernel
        hipDeviceSynchronize();

        // Normalize the matrix
        normalize_kernel<<<numBlocks, threadsPerBlock>>>(d_Mat_A, d_Mat_Stencil, d_FNorm, N, M);
        hipDeviceSynchronize();

        hipMemcpy(Mat_A.data(), d_Mat_A, N * M * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&FNorm, d_FNorm, sizeof(float), hipMemcpyDeviceToHost);
        unsigned long long end = rdtsc();

        /*for(int i=0;i<N*M;i+=M)
        {
            for(int j=i;j<(i+M);j++)
                printf("%f ",Mat_A[j]);
            printf("\n");
        }*/

        elapsed_time[count] = (double)(end - start) / ClkPerSec;
        printf("TTC : %.12f\n", elapsed_time[count]);
    }


    /*for(int i=0;i<N*M;i+=M)
    {
        for(int j=i;j<(i+M);j++)
            printf("%f ",Mat_A[j]);
        printf("\n");
    }*/ 

    // Print average time
    for (int count = 1; count < loops; count++) {
        Average += elapsed_time[count];
    }
    Average /= (loops - 1);
    printf("Average Computation Time: %.12f\n", Average);

    hipFree(d_Mat_A);
    hipFree(d_Mat_Stencil);
    hipFree(d_FNorm);

    return 0;
}
